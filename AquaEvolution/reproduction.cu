#include "hip/hip_runtime.h"
#include <simulation/structs/reproduction.cuh>

#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/sort.h>
#include <thrust/random.h>

struct ChildrenCountFunctorAlgae {
	__device__
	int32_t operator()(const Algae::Entity& e) const {
		auto is_alive = e.get<2>();
		if (!is_alive) return 0;

		auto energy = e.get<3>();

		int32_t children = fmaxf(energy - Algae::ENERGY_MINIMUM_TO_REPRODUCT, 0.f) / Algae::ENERGY_PER_KID;
		return children;
	}
};

struct ChildrenCountFunctorFish {
	__device__
		int32_t operator()(const Fish::Entity& e) const {
		auto is_alive = e.get<2>();
		if (!is_alive) return 0;

		auto energy = e.get<3>();

		int32_t children = fmaxf(energy - Fish::ENERGY_MINIMUM_TO_REPRODUCT, 0.f) / Fish::ENERGY_PER_KID;
		return children;
	}
};

struct ChildrenPerIterFunctor {
	__device__ int32_t operator()(const int32_t& x) const {
		return int32_t(x > 0);
	}
};

struct Decrement {
	__device__ void operator()(int32_t& x) {
		x--;
	}
};

struct GeneratorAlgae {
	using tup = thrust::tuple<float2, uint32_t>;
	__device__
		Algae::Entity operator()(const tup& n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float> dist(-1, 1);
		rng.discard(n.get<1>());

		float2 pos = n.get<0>();
		float2 vec = normalize(make_float2(dist(rng), dist(rng)));
		bool alive = true;
		float currentEnergy = Algae::INIT_ENERGY;
		return thrust::make_tuple(pos, vec, alive, currentEnergy);
	}
};

struct GeneratorFish {
	using tup = thrust::tuple<Fish::Entity, uint32_t>;
	__device__
		Fish::Entity operator()(const tup& n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float> dist(-1, 1);
		rng.discard(n.get<1>());

		auto fishEntity = n.get<0>();
		float2 pos = fishEntity.get<0>();
		float2 vec = normalize(make_float2(dist(rng), dist(rng)));
		float2 energyParams = fishEntity.get<6>();
		float2 sightParams = fishEntity.get<7>();
		float velocity = fishEntity.get<8>();
		bool alive = true;
		float currentEnergy = fminf(Fish::INITAL_ENERGY, energyParams.x);
		FishDecisionEnum next = FishDecisionEnum::NONE;
		uint64_t eatenAlgaeId = -1;

		//traits and mutations
		// TO DO

		return thrust::make_tuple(pos, vec, alive, currentEnergy, next, eatenAlgaeId,energyParams,sightParams,velocity);
	}
};

void Aquarium::reproduction_algae() {

	thrust::device_vector<int32_t> dc(algae->device.positions.size());
	auto it = algae->device.iter();
	
	thrust::transform(thrust::device, it.get<0>(), it.get<1>(), dc.begin(), ChildrenCountFunctorAlgae());

	//thrust::host_vector<int32_t> h = dc;
	//for (int32_t i = 0; i < h.size(); i++) {
	//	std::cout << i << ": " << h[i] << std::endl;
	//}


	auto begin = thrust::make_zip_iterator(thrust::make_tuple(
		algae->device.positions.begin(),
		dc.begin()
	));
	thrust::sort_by_key(thrust::device, dc.begin(), dc.end(), begin, thrust::greater<int32_t>());

	int32_t childrenLeft = thrust::reduce(dc.begin(), dc.end());
	int32_t minLeft = 0;
	int32_t childrenInLoop = thrust::transform_reduce(
		dc.begin(), dc.end(), ChildrenPerIterFunctor(), 0, thrust::plus<int32_t>());
	std::cout << "reduction: " << childrenLeft << std::endl;

	Algae* back = &algaeBuffer[1 - currentAlgaeBuffer];
	if (childrenLeft > back->capacity) {
		minLeft = childrenLeft - back->capacity;
	}
	
	back->resize(back->device, childrenLeft - minLeft);

	auto& backIter = back->device.iter().get_head();
	auto countIter = thrust::make_counting_iterator<uint32_t>(0);
	while (childrenLeft > minLeft) {

		if (childrenInLoop > childrenLeft - minLeft)
			childrenInLoop = childrenLeft - minLeft;

		thrust::transform(
			thrust::make_zip_iterator(thrust::make_tuple(algae->device.positions.begin(), countIter)),
			thrust::make_zip_iterator(thrust::make_tuple(algae->device.positions.begin() + childrenInLoop, countIter + childrenInLoop)),
			backIter,
			GeneratorAlgae()
		);

		backIter += childrenInLoop;
		
		thrust::for_each(dc.begin(), dc.end(), Decrement());
		childrenLeft = thrust::reduce(dc.begin(), dc.end());
		childrenInLoop = thrust::transform_reduce(
			dc.begin(), dc.end(), ChildrenPerIterFunctor(), 0, thrust::plus<int32_t>());
	}

	algae = back;
	currentAlgaeBuffer = 1 - currentAlgaeBuffer;
}

void Aquarium::reproduction_fish()
{
	thrust::device_vector<int32_t> dc(fish->device.positions.size());
	auto it = fish->device.iter();

	thrust::transform(thrust::device, it.get<0>(), it.get<1>(), dc.begin(), ChildrenCountFunctorFish());

	//thrust::host_vector<int32_t> h = dc;
	//for (int32_t i = 0; i < h.size(); i++) {
	//	std::cout << i << ": " << h[i] << std::endl;
	//}


	auto begin = thrust::make_zip_iterator(thrust::make_tuple(
		fish->device.positions.begin(),
		dc.begin()
	));
	thrust::sort_by_key(thrust::device, dc.begin(), dc.end(), begin, thrust::greater<int32_t>());

	int32_t childrenLeft = thrust::reduce(dc.begin(), dc.end());
	int32_t minLeft = 0;
	int32_t childrenInLoop = thrust::transform_reduce(
		dc.begin(), dc.end(), ChildrenPerIterFunctor(), 0, thrust::plus<int32_t>());
	std::cout << "fish reduction: " << childrenLeft << std::endl;

	Fish* back = &fishBuffer[1 - currentFishBuffer];
	if (childrenLeft > back->capacity) {
		minLeft = childrenLeft - back->capacity;
	}

	back->resize(back->device, childrenLeft - minLeft);

	auto& backIter = back->device.iter().get_head();
	auto countIter = thrust::make_counting_iterator<uint32_t>(0);
	while (childrenLeft > minLeft) {

		if (childrenInLoop > childrenLeft - minLeft)
			childrenInLoop = childrenLeft - minLeft;

		auto it = fish->device.iter();
		thrust::transform(
			thrust::make_zip_iterator(thrust::make_tuple(it.get_head(), countIter)),
			thrust::make_zip_iterator(thrust::make_tuple(it.get_head() + childrenInLoop, countIter + childrenInLoop)),
			backIter,
			GeneratorFish()
		);

		backIter += childrenInLoop;

		thrust::for_each(dc.begin(), dc.end(), Decrement());
		childrenLeft = thrust::reduce(dc.begin(), dc.end());
		childrenInLoop = thrust::transform_reduce(
			dc.begin(), dc.end(), ChildrenPerIterFunctor(), 0, thrust::plus<int32_t>());
	}

	fish = back;
	currentFishBuffer = 1 - currentFishBuffer;
}
